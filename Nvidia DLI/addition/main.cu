
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

using namespace std;

__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}

int main() {
    int N = 1<<20; // this is 1 million elements wtf
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    //initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    //run kernel on 1m elements on the cpu
    add<<<numBlocks, blockSize>>>(N, x, y);

    //wait for gpu to finish before accessing on host
    hipDeviceSynchronize();

    //check for errors all vals shd be 3.0f
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    cout << "Max error: " << maxError << endl;


    hipFree(x);
    hipFree(y);
    return 0;
}